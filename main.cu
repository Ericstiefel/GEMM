#include "hip/hip_runtime.h"
#include "gemm.h"
#include <iostream>
#include <vector>
#include <random>
#include <ctime>
#include <chrono> 
#include <hip/hip_runtime.h>
#include <cmath> 

// --- Configuration ---
const int M = 1024; // Rows of A and C
const int K = 1024; // Cols of A and Rows of B
const int N = 1024; // Cols of B and C
const int TILE_WIDTH = 32; // Tile width for the tiled GPU implementation

// --- CUDA Error Checking Macro ---
#define CUDA_CHECK(err) { \
    hipError_t err_ = (err); \
    if (err_ != hipSuccess) { \
        std::cerr << "CUDA error in " << __FILE__ << " at line " << __LINE__ \
                  << ": " << hipGetErrorString(err_) << std::endl; \
        exit(EXIT_FAILURE); \
    } \
}

// --- Helper function to initialize matrices ---
void initializeMatrix(float* mat, int rows, int cols) {
    std::mt19937 mt(static_cast<unsigned int>(time(0)));
    std::uniform_real_distribution<float> dist(0.0f, 1.0f);
    for (int i = 0; i < rows * cols; ++i) {
        mat[i] = dist(mt);
    }
}

// --- Helper function to verify results ---
void verify_result(const float* cpu_res, const float* gpu_res, int m, int n) {
    const float tolerance = 1e-5;
    for (int i = 0; i < m * n; ++i) {
        if (fabs(cpu_res[i] - gpu_res[i]) > tolerance) {
            std::cerr << "Verification FAILED at index " << i
                      << "! CPU: " << cpu_res[i]
                      << ", GPU: " << gpu_res[i] << std::endl;
            return;
        }
    }
    std::cout << "Verification PASSED!" << std::endl;
}

int main() {
    // --- Host Memory Allocation ---
    float *h_A, *h_B, *h_C_cpu, *h_C_gpu_naive, *h_C_gpu_tiled;
    h_A = new float[M * K];
    h_B = new float[K * N];
    h_C_cpu = new float[M * N];
    h_C_gpu_naive = new float[M * N];
    h_C_gpu_tiled = new float[M * N];

    // --- Initialize Host Matrices ---
    initializeMatrix(h_A, M, K);
    initializeMatrix(h_B, K, N);
    std::cout << "Matrices (" << M << "x" << K << " and "
              << K << "x" << N << ") generated successfully." << std::endl;

    // --- Device Memory Allocation ---
    float *d_A, *d_B, *d_C;
    CUDA_CHECK(hipMalloc(&d_A, M * K * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_B, K * N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_C, M * N * sizeof(float)));

    // --- Copy Data from Host to Device ---
    CUDA_CHECK(hipMemcpy(d_A, h_A, M * K * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, h_B, K * N * sizeof(float), hipMemcpyHostToDevice));

    // --- Profiling Events ---
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    float gpu_time = 0.0f;

    // =======================================================
    // 1. Profile CPU Implementation
    // =======================================================
    std::cout << "\nRunning CPU implementation..." << std::endl;
    auto cpu_start = std::chrono::high_resolution_clock::now();
    cpu_gemm(h_A, h_B, h_C_cpu, M, K, N);
    auto cpu_end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> cpu_duration = cpu_end - cpu_start;
    std::cout << "CPU Time: " << cpu_duration.count() << " ms" << std::endl;

    // =======================================================
    // 2. Profile Naive GPU Implementation (BASELINE)
    // =======================================================
    std::cout << "\nRunning Naive GPU implementation (Baseline)..." << std::endl;
    CUDA_CHECK(hipEventRecord(start));
    naive_launcher(d_A, d_B, d_C, M, K, N);
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&gpu_time, start, stop));
    std::cout << "Naive GPU Time: " << gpu_time << " ms" << std::endl;
    CUDA_CHECK(hipMemcpy(h_C_gpu_naive, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost));
    verify_result(h_C_cpu, h_C_gpu_naive, M, N);

    // =======================================================
    // 3. Profile Tiled GPU Implementation (COMPARISON)
    // =======================================================
    std::cout << "\nRunning Tiled (Double Buffered) GPU implementation (Comparison)..." << std::endl;
    CUDA_CHECK(hipMemset(d_C, 0, M * N * sizeof(float))); // Clear device memory for C
    CUDA_CHECK(hipEventRecord(start));
    tiled_buff_launcher<TILE_WIDTH>(d_A, d_B, d_C, M, K, N);
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&gpu_time, start, stop));
    std::cout << "Tiled GPU Time: " << gpu_time << " ms" << std::endl;
    CUDA_CHECK(hipMemcpy(h_C_gpu_tiled, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost));
    verify_result(h_C_cpu, h_C_gpu_tiled, M, N);

    // --- Cleanup ---
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));
    delete[] h_A;
    delete[] h_B;
    delete[] h_C_cpu;
    delete[] h_C_gpu_naive;
    delete[] h_C_gpu_tiled;

    return 0;
}
